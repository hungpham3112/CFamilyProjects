#include "hip/hip_runtime.h"
#include "./kernel.h"

// CUDA kernel optimized 1

__global__ void CopyBufferOpt1(const unsigned int* src, unsigned int* dst, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int coarseningFactor = 4;
    int coarsenedId = id * coarseningFactor;
    if (id < size) {
        for (int i = 0; i < coarseningFactor; i++) {
            dst[coarsenedId + i] = src[coarsenedId + i];
        }
    }
}

int main(int argc, char *argv[])
{
        
    size_t size = ARR_LEN * sizeof(unsigned int);
    unsigned int* h_A = (unsigned int*)malloc(size);
    unsigned int* h_B = (unsigned int*)malloc(size);
    generateRandomUnsignedIntArray(h_A, ARR_LEN, 0, 1000);
    // ShowArr(h_A, size, "h_A");
    unsigned int *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void** )&d_B, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int threadPerBlock = 256;
    int blockPerGrid = (ARR_LEN + threadPerBlock - 1) / threadPerBlock;
    CopyBufferOpt1<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, ARR_LEN);

    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    performApproxTest(h_A, h_B, ARR_LEN);
    performIdenticalTest(h_A, h_B, ARR_LEN);
    // ShowArr(h_B, size, "h_B");
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);
    return 0;
}
