#include "hip/hip_runtime.h"
#include "./kernel.h"

// CUDA kernel optimized 2

__global__ void CopyBufferOpt2(const unsigned int* src, unsigned int* dst) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < 512) {
        dst[id] = src[id];
    } else {
        dst[id] = 0;
    }
}

__global__ void showBlockIndices() {
    // Access block indices
    int blockIndexX = blockIdx.x;
    int blockIndexY = blockIdx.y;
    int blockIndexZ = blockIdx.z;

    // Print block indices
    printf("BlockIdx: (%d, %d, %d)\n", blockIndexX, blockIndexY, blockIndexZ);
}

int main(int argc, char *argv[])
{
        
    size_t size = ARR_LEN * sizeof(unsigned int);
    unsigned int* h_A = (unsigned int*)malloc(size);
    unsigned int* h_B = (unsigned int*)malloc(size);
    generateRandomUnsignedIntArray(h_A, ARR_LEN, 0, 1000);
    // ShowArr(h_A, size, "h_A");
    unsigned int *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void** )&d_B, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int threadPerBlock = 512;
    int blockPerGrid = (ARR_LEN + threadPerBlock - 1) / threadPerBlock;
    CopyBufferOpt2<<<blockPerGrid, threadPerBlock>>>(d_A, d_B);
    // showBlockIndices<<<blockPerGrid, threadPerBlock>>>();

    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    // performApproxTest(h_A, h_B, ARR_LEN);
    // performIdenticalTest(h_A, h_B, ARR_LEN);
    // ShowArr(h_B, ARR_LEN, "h_B");
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);
    return 0;
}
