#include "hip/hip_runtime.h"
// CUDA Kernel unoptimized
#include "./kernel.h"

__global__ void CopyBufferOrigin(unsigned int* src, unsigned int* dst, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < size) {
        dst[id] = src[id];
    }
}


int main(int argc, char *argv[])
{
        
    size_t size = ARR_LEN * sizeof(unsigned int);
    unsigned int* h_A = (unsigned int*)malloc(size);
    unsigned int* h_B = (unsigned int*)malloc(size);
    generateRandomUnsignedIntArray(h_A, ARR_LEN, 0, 100);
    // ShowArr(h_A, size, "h_A");
    unsigned int *d_A, *d_B;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void** )&d_B, size);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    int threadPerBlock = 256;
    int blockPerGrid = (ARR_LEN + threadPerBlock - 1) / threadPerBlock;
    CopyBufferOrigin<<<blockPerGrid, threadPerBlock>>>(d_A, d_B, size);

    hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    performTest(h_A, h_B, ARR_LEN);
    // ShowArr(h_B, size, "h_B");
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);
    return 0;
}
