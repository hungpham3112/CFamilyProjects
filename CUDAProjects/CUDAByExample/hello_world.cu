#include "hip/hip_runtime.h"
#include <iostream>

__global__ vecAddKernel(float* A, float* B, float* C, int vec_len) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < vec_len) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ vecAdd(float* A_h, float* B_h, float* C_h, int vec_len) {
    int size = n * sizeof(float);
    float* A_d, B_d, C_d;
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyDeviceToHost);
    hipMemcpy(B_d, B_h, size, hipMemcpyDeviceToHost);

    int dimBlock = 256
    int dimGrid = ceil(n / 256.0);
    vecAddKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, vec_len);

    hipMemcpy(C_h, C_d, size, hipMemcpyHostToDevice);
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

}
int main(int argc, char *argv[])
{
    
    return 0;
}



