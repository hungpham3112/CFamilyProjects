#include "hip/hip_runtime.h"
#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <stdio.h>
#include <cadna.h>
#include <cadna_gpu.h>

template <class T>
struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

// specialize for double to avoid unaligned memory
// access compile errors
template <>
struct SharedMemory<double> {
  __device__ inline operator double *() {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }

  __device__ inline operator const double *() const {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }
};

template <class T>
__device__ __forceinline__ T warpReduceSum(unsigned int mask, T mySum) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    mySum += __shfl_down_sync(mask, mySum, offset);
  }
  return mySum;
}

#if __CUDA_ARCH__ >= 800
// Specialize warpReduceFunc for int inputs to use __reduce_add_sync intrinsic
// when on SM 8.0 or higher
template <>
__device__ __forceinline__ int warpReduceSum<int>(unsigned int mask,
                                                  int mySum) {
  mySum = __reduce_add_sync(mask, mySum);
  return mySum;
}
#endif

template <class T>
__global__ void reduce0(T *g_idata, T *g_odata, unsigned int n) {
  cadna_init_gpu();
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();

  // load shared mem
  T *sdata = SharedMemory<T>();

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    sdata[tid] = g_idata[i];
  } else {
    sdata[tid] = static_cast<T>(0.0);
  }

  cg::sync(cta);
  // do reduction in shared mem

  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    // modulo arithmetic is slow!
    if ((tid % (2 * s)) == 0) {
      sdata[tid] = sdata[tid] + sdata[tid + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0)  g_odata[blockIdx.x] = sdata[0]; 
}
