
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <sys/time.h>


#ifndef N
#define N 0x1000000
//#define N 4
#endif

#ifndef TAILLE_BLOC_X
#define TAILLE_BLOC_X 160
#endif

#ifndef NLOOP
#define NLOOP 1
#endif

#ifndef CPT
#define CPT 1
#endif

#ifdef DOUBLE
#ifdef CADNA
#include <cadna.h>
#include "cadna_gpu.cu"
#define DATATYPE double
#define DATATYPECADNA double_st
#define DATATYPECADNAGPU double_gpu_st
#else
#define DATATYPE double
#define DATATYPECADNA double
#define DATATYPECADNAGPU double
#endif
#elif FLOAT
#ifdef CADNA
#include <cadna.h>
#include "cadna_gpu.cu"
#define DATATYPE float
#define DATATYPECADNA float_st
#define DATATYPECADNAGPU float_gpu_st
#else
#define DATATYPE float
#define DATATYPECADNA float
#define DATATYPECADNAGPU float
#endif
#else
#ifdef CADNA
#include <cadna.h>
#include "cadna_gpu.cu"
//#include "cadna_gpu_half2.cu"
//#include "cadna_gpu_float.h"
//#include "cadna_gpu_float.cu"
#define DATATYPE float
#define DATATYPECADNA float_st
#define DATATYPECADNAGPU half2_gpu_st
#define DATATYPECADNAGPUF float_gpu_st
#else
#include <hip/hip_fp16.h>
#define DATATYPE float
#define DATATYPECADNA float
#define DATATYPECADNAGPU half2
#define DATATYPECADNAGPUF float
#endif
#endif

#define SEED 1


using namespace std;

// Génération aléatoire du tableau :
__host__ void aleaTabSt(DATATYPECADNA* T, int n){
  long int i;
  srand(SEED);

  for(i = 0; i < n; i++){
#ifdef CADNA
    T[i] = DATATYPECADNA(1.0f + 0.01f*((DATATYPE)rand()/(DATATYPE)RAND_MAX));
#else
    T[i] = 1.0f + 0.01f*((DATATYPE)rand()/(DATATYPE)RAND_MAX);
#endif
  }
}

// Mesures :
double my_gettimeofday(){
  struct timeval tmp_time;
  gettimeofday(&tmp_time, NULL);
  return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}

__host__ void compute(DATATYPECADNA *a, DATATYPECADNA *b){
  for (int i = 0; i < N/2; i++){
    DATATYPECADNA f = a[i];
    DATATYPECADNA g = b[i];
    for (int j = 0; j < NLOOP; j++)
      f = g * f;
    a[i] = f;
  }
}

__global__ void computeGPU(DATATYPECADNAGPUF *a0, DATATYPECADNAGPUF *a1, DATATYPECADNAGPUF *b0, DATATYPECADNAGPUF *b1){
  int i = blockDim.x * blockIdx.x + threadIdx.x ;
#ifdef CADNA
  cadna_init_gpu();

  if(i<N){
    DATATYPECADNAGPU f = __float_gpu_st2half2_gpu_st(a0[i], a1[i]);
    DATATYPECADNAGPU g = __float_gpu_st2half2_gpu_st(b0[i], b1[i]);
    for(int j=0; j<NLOOP; j++){
      f = g * f;
    }
    a0[i] = __half2_gpu_st2float_gpu_st_low(f);
    a1[i] = __half2_gpu_st2float_gpu_st_high(f);
  }
#else

  if(i<N){
    DATATYPECADNAGPU f = __floats2half2_rn(a0[i], a1[i]);
    DATATYPECADNAGPU g = __floats2half2_rn(b0[i], b1[i]);
    for(int j=0; j<NLOOP; j++){
      //f = g * f;
      f = __hmul2(g , f);
    }
    a0[i] = __low2float(f);
    a1[i] = __high2float(f);
  }
#endif
}


/*
__global__ void computeGPU(DATATYPECADNAGPUF *a0, DATATYPECADNAGPUF *a1, DATATYPECADNAGPUF *b0, DATATYPECADNAGPUF *b1){
  int i = blockDim.x * blockIdx.x + threadIdx.x ;
#ifdef CADNA
  cadna_init_gpu();

  if(i<N){
    DATATYPECADNAGPU f = __float_gpu_st2half2_gpu_st(a0[i], a1[i]);
    DATATYPECADNAGPU g = __float_gpu_st2half2_gpu_st(b0[i], b1[i+1]);

    a0[i] = __half2_gpu_st2float_gpu_st_low(f);
    a1[i] = __half2_gpu_st2float_gpu_st_high(f);
  }
#else

  if(i<N){
    DATATYPECADNAGPU f = __floats2half2_rn(a0[i], a1[i]);
    DATATYPECADNAGPU g = __floats2half2_rn(b0[i], b1[i+1]);
  
    a0[i] = __low2float(f);
    a1[i] = __high2float(f);
  }
#endif
}
*/





int main(int argc, char **argv){

  hipError_t sync_error;
#ifdef CADNA
  cadna_init(-1, CADNA_INTRINSIC | CADNA_CANCEL);
#endif
  int ic;
#ifndef CADNA
  long int taille_totale = (N)*sizeof(DATATYPE);
#else
  long int taille_totale = (N)*sizeof(DATATYPECADNA);
#endif

  double t_startGPU, t_endGPU;
  double t_GPU;

  DATATYPECADNAGPUF *d_a0, *d_a1, *d_b0, *d_b1;

  DATATYPECADNA *a0 = (DATATYPECADNA*)malloc(taille_totale);
  DATATYPECADNA *a1 = (DATATYPECADNA*)malloc(taille_totale);
  DATATYPECADNA *b0 = (DATATYPECADNA*)malloc(taille_totale);
  DATATYPECADNA *b1 = (DATATYPECADNA*)malloc(taille_totale);
  DATATYPECADNA *a0_GPU = (DATATYPECADNA*)malloc(taille_totale);
  DATATYPECADNA *a1_GPU = (DATATYPECADNA*)malloc(taille_totale);

  aleaTabSt(a0, N);
  aleaTabSt(a1, N);
  aleaTabSt(b0, N);
  aleaTabSt(b1, N);

  /* Allocation GPU : */
  hipMalloc((void **) &d_a0, taille_totale);
  hipMalloc((void **) &d_a1, taille_totale);
  hipMalloc((void **) &d_b0, taille_totale);
  hipMalloc((void **) &d_b1, taille_totale);

  /* Transferts CPU -> GPU (synchrones) : */
  hipMemcpy(d_a0, a0, taille_totale, hipMemcpyHostToDevice);
  hipMemcpy(d_a1, a1, taille_totale, hipMemcpyHostToDevice);
  hipMemcpy(d_b0, b0, taille_totale, hipMemcpyHostToDevice);
  hipMemcpy(d_b1, b1, taille_totale, hipMemcpyHostToDevice);


  /* Lancement de kernel (asynchrone) : */
  dim3 threadsParBloc(TAILLE_BLOC_X);
  dim3 tailleGrille(ceil(N/(float) TAILLE_BLOC_X)); // Nombre de blocs


  t_startGPU = my_gettimeofday();
  for(ic=0; ic<CPT;ic++){
    computeGPU<<<tailleGrille, threadsParBloc>>>(d_a0, d_a1, d_b0, d_b1);
    hipDeviceSynchronize();
  }
  t_endGPU =  my_gettimeofday();

  sync_error = hipGetLastError();
  if(sync_error != hipSuccess) {
    fprintf(stderr, "[CUDA SYNC ERROR at %s:%d -> %s]\n",
	    __FILE__ , __LINE__, hipGetErrorString(sync_error));
    exit(EXIT_FAILURE);
  }

#ifdef NUMCHECK
  double t_startCPU, t_endCPU;
  t_startCPU = my_gettimeofday();
  for(ic=0; ic<CPT;ic++){
    compute(a0,b0);

  }
  t_endCPU = my_gettimeofday();
#endif

  hipMemcpy(a0_GPU, d_a0, taille_totale, hipMemcpyDeviceToHost);
  hipMemcpy(a1_GPU, d_a1, taille_totale, hipMemcpyDeviceToHost);

  t_GPU = (t_endGPU - t_startGPU)/CPT;
#ifdef NUMCHECK
  double t_CPU;
  t_CPU = (t_endCPU - t_startCPU)/CPT;

  DATATYPECADNA max, moy;

  max = (DATATYPE)0.f;
  moy = (DATATYPE)0.f;
  int i;
  for(i = 0; i < N; i++) {
    if (a0[i] != (DATATYPE)0.f) {
      DATATYPECADNA relerr = fabsf((a0[i]-a0_GPU[i])/a0[i]);
      moy = moy + relerr;
      if (relerr > max)
	max = relerr;
    }
  }

  moy = moy / (DATATYPE)N;

  cerr << TAILLE_BLOC_X << " " << max << " " << moy << " " << t_GPU << " " << t_CPU << endl;
#else
  cerr << TAILLE_BLOC_X << " " << t_GPU << endl;
#endif

#ifdef CADNA
  cadna_end();
#endif
  /* Libération mémoire GPU et CPU : */
  hipFree(d_a0); hipFree(d_b0);
  hipFree(d_a1); hipFree(d_b1);
  free(a0); free(b0); free(a0_GPU);
  free(a1); free(b1); free(a1_GPU);
  // fclose(fic);
  return EXIT_SUCCESS;
}
