
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <sys/time.h>
#include "hip/hip_fp16.h"


#ifndef N
//#define N 0x10000000
#define N 4
#endif

#ifndef TAILLE_BLOC_X
#define TAILLE_BLOC_X 32
#endif

#ifndef NLOOP
#define NLOOP 1
#endif

#ifndef CPT
#define CPT 1
#endif


// CADNA
#ifdef CADNA
#include <cadna.h>
#include "cadna_gpu.cu"
#define DATATYPE half
#define DATATYPECADNA float_st
#define DATATYPECADNAGPU half_gpu_st
#else
#define DATATYPE half
#define DATATYPECADNA half
#define DATATYPECADNAGPU half
#endif



#define SEED 1

using namespace std;

// Génération aléatoire du tableau :
__host__ void aleaTabSt(float* T, int n){
    long int i;
    srand(SEED);
    //    posix_memalign((void **)&T, 64, N * sizeof(float));
    for(i = 0; i < n; i++){
        T[i] =(float)(0.5f + (float)rand()/(float)RAND_MAX);
    }
}

// Conversion :
__device__ void convertIntoSt(DATATYPE * T, DATATYPECADNA * Tst, int n){
     for(int i = 0; i < n; i++){
        Tst[i] = T[i];
    }
}

// Mesures :
double my_gettimeofday(){
    struct timeval tmp_time;
    gettimeofday(&tmp_time, NULL);
    return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}

/*
__host__ void compute(float *a, float *b){
    for (int i = 0; i < N; i++){
        float f = a[i];
        for (int j = 0; j < NLOOP; j++)
            f = b[i] + f;
        a[i] = f;
    }
}
*/
__global__ void computeGPU(float *a, float *b){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i<N){
        half f = __float2half(a[i]);
        for(int j=0; j<NLOOP; j++){
            f = __hadd(__float2half(b[i]), f);
        }
        a[i] = __half2float(f);
    }
}




int main(int argc, char **argv){
    int i, ic;
    int taille_totale = N*sizeof(half);

#ifdef DEBUG
    int nb_oper =0;
#endif
    //double t_startCPU, t_endCPU;
    double t_startGPU, t_endGPU;


    float *d_a, *d_b;

    float *a = (float*)malloc(taille_totale);
    float *b = (float*)malloc(taille_totale);
    aleaTabSt(a, N);
    aleaTabSt(b, N);



 	float *a_GPU = (float*)malloc(taille_totale);

    // Concatène par défaut, écrase si nouvelle série.
    //FILE* fic=(TAILLE_BLOC_X == 32)?fopen("operBXaddComb.res", "w"):fopen("operBXaddComb.res", "a");



   // int valide = 1;




    /* Allocation GPU : */
    hipMalloc((void **) &d_a, taille_totale);
    hipMalloc((void **) &d_b, taille_totale);

    // Transferts CPU -> GPU (synchrones) :
    // Passage en DATATYPECADNAGPU implicite.
    hipMemcpy(d_a, a, taille_totale, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, taille_totale, hipMemcpyHostToDevice);


    /* Lancement de kernel (asynchrone) : */
    dim3 threadsParBloc(TAILLE_BLOC_X);
    dim3 tailleGrille(ceil(N/(float) TAILLE_BLOC_X)); // Nombre de blocs



    t_startGPU = my_gettimeofday();
    for(ic=0; ic<CPT;ic++){
        computeGPU<<<tailleGrille, threadsParBloc>>>(d_a,d_b);
        hipDeviceSynchronize();
    }
    t_endGPU =  my_gettimeofday();



    /*t_startCPU = my_gettimeofday();
    for(ic=0; ic<CPT;ic++){
        compute(a,b);
    }
    t_endCPU = my_gettimeofday();
*/

    hipMemcpy(a_GPU, d_a, taille_totale, hipMemcpyDeviceToHost);


    /*for(i = 0; i < 5; i++)
        cerr << "Result[" << i << "] :" << a[i] << endl;

    cout << "Time: " << t_endCPU-t_startCPU << endl;
*/
    for(i = 0; i < 5; i++)
        cerr << "Result[" << i << "] :" << a[i] << endl;

    cout << "Time: " << t_endGPU-t_startGPU << endl;


  //  t_CPU = (t_endCPU - t_startCPU)/CPT;
   // t_GPU = (t_endGPU - t_startGPU)/CPT;
/*
    for(i = 0; i < 5; i++)
        if (a[i] != a_GPU[i]){
            valide = 0;
            cout<< TAILLE_BLOC_X << " " << valide << " " << t_CPU << " " << t_GPU << endl;
            fprintf(fic, "%d %d %f %f %d %d\n", TAILLE_BLOC_X, valide, t_CPU, t_GPU, N, NLOOP, CPT);
            exit(1);
        }
*/
  //  fprintf(fic, "%d %d %f %f %d %d\n", TAILLE_BLOC_X, valide, t_CPU, t_GPU, N, NLOOP, CPT);
    //cout<< TAILLE_BLOC_X << " " << valide << " " << t_CPU << " " << t_GPU << endl;

    /* Libération mémoire GPU et CPU : */
    //cudaFree(d_a); cudaFree(d_b);
    //free(a); free(b); free(a_GPU);


    //fclose(fic);
    return EXIT_SUCCESS;
}
