#include "hip/hip_runtime.h"
// Copyright 2019   J.-M. Chesneaux, P. Eberhart, F. Jezequel, J.-L. Lamotte, S. Zhou

// This file is part of CADNA.

// CADNA is free software: you can redistribute it and/or modify
// it under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// CADNA is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU Lesser General Public License for more details.

// You should have received a copy of the GNU Lesser General Public License
// along with CADNA.  If not, see <http://www.gnu.org/licenses/>.
// variables globales
# include "cadna_gpu_float.h"
#include "cadna_gpu_double.h"

#ifdef HALF


#elif HALF2

#else
#define MAX_THREAD_PER_BLOCK 1024 // 512

//#define MAX_BLOCK_SIZE_X 1024// 512

#define MAX_THREAD (gridDim.x*blockDim.x * gridDim.y*blockDim.y * gridDim.z*blockDim.z)

#define CUDA_ERROR(cuda_call) {					\
    hipError_t error = cuda_call;				\
    if(error != hipSuccess){					\
      fprintf(stderr, "[CUDA ERROR at %s:%d -> %s]\n",		\
	      __FILE__ , __LINE__, hipGetErrorString(error));  \
      exit(EXIT_FAILURE);					\
    }								\
  }

__device__ unsigned int _cadna_TauswortheStep(unsigned int, unsigned int,
					      unsigned int,
					      unsigned int,
					      unsigned int) __attribute__((always_inline));
__device__ inline unsigned int _cadna_TauswortheStep(unsigned int seed, unsigned int s1,
						     unsigned int s2,
						     unsigned int s3,
						     unsigned int m){
  unsigned int b = (((seed << s1) ^ seed) >> s2);
  return  (((seed & m) << s3) ^ b);
}

__device__ inline unsigned int _cadna_LCGStep(unsigned int, unsigned int,
					      unsigned int) __attribute__((always_inline));
__device__ inline unsigned int _cadna_LCGStep(unsigned int seed, unsigned int a,
					      unsigned int c){
  return (a * seed + c);
}

__device__ __shared__ unsigned int seed[MAX_THREAD_PER_BLOCK];
__device__ __shared__ unsigned char _cadna_random_counter[MAX_THREAD_PER_BLOCK];

__device__ void cadna_init_gpu()
{
  unsigned int idx, init;

  idx = // (gridDim.x*blockDim.x * gridDim.y*blockDim.y) * blockIdx.z +
    // (gridDim.x*blockDim.x) * blockIdx.y +
    // blockDim.x * blockIdx.x +
    (blockDim.x*blockDim.y) * threadIdx.z +
    blockDim.x * threadIdx.y +
    threadIdx.x ;
  init = (gridDim.x*blockDim.x * gridDim.y*blockDim.y) * blockIdx.z +
    (gridDim.x*blockDim.x) * blockIdx.y +
    blockDim.x * blockIdx.x +
    (blockDim.x*blockDim.y) * threadIdx.z +
    blockDim.x * threadIdx.y +
    threadIdx.x ;

  seed[idx] = init*1099087573UL;
  _cadna_random_counter[idx] = 0;
  
  // __syncthreads();
}

__device__ inline  unsigned int RANDOMGPU()
{

    unsigned int idx;
    idx = // (gridDim.x*blockDim.x * gridDim.y*blockDim.y) * blockIdx.z +
      // (gridDim.x*blockDim.x) * blockIdx.y +
      // blockDim.x * blockIdx.x +
      (blockDim.x*blockDim.y) * threadIdx.z +
      blockDim.x * threadIdx.y +
      threadIdx.x ;

    // /!\ N'est plus static sur GPU
    const unsigned int Taus1S1 = 13;
    const unsigned int Taus1S2 = 19;
    const unsigned int Taus1S3 = 12;
    const unsigned int Taus1M  = 429496729U;
    const unsigned int Taus2S1 = 2;
    const unsigned int Taus2S2 = 25;
    const unsigned int Taus2S3 = 4;
    const unsigned int Taus2M  = 4294967288U;
    const unsigned int Taus3S1 = 3;
    const unsigned int Taus3S2 = 11;
    const unsigned int Taus3S3 = 17;
    const unsigned int Taus3M  = 429496280U;
    const unsigned int LCGa    = 1664525;
    const unsigned int LCGc    = 1013904223U;

    unsigned int z1, z2, z3, z4;

    // À supposer qu'il est peut coûteux de modifier seed. On économise un tableau de _cadna_random.
    // Test : si le compteur est à 0 ou 32 :
    if((_cadna_random_counter[idx]&0xF)==0){
        _cadna_random_counter[idx] = 0;

        z1 = _cadna_TauswortheStep(seed[idx], Taus1S1, Taus1S2, Taus1S3, Taus1M);
        z2 = _cadna_TauswortheStep(seed[idx], Taus2S1, Taus2S2, Taus2S3, Taus2M);
        z3 = _cadna_TauswortheStep(seed[idx], Taus3S1, Taus3S2, Taus3S3, Taus3M);
        z4 = _cadna_LCGStep(seed[idx], LCGa, LCGc);
        seed[idx] = (z1^ z2 ^ z3 ^ z4);
    }

    return seed[idx]>>((_cadna_random_counter[idx]++)*2)&3;
}
#endif

/////////////////////////////////////////////////////

__device__ float_gpu_st operator+(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;

  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,b.x);
  else res.x=__fadd_rd(a.x,b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,b.y);
    res.z=__fadd_rd(a.z,b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,b.y);
    res.z=__fadd_ru(a.z,b.z);
  }

  res.error=a.error | b.error;

  return res;
}


__device__ float_gpu_st operator+(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;

  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,b);
  else res.x=__fadd_rd(a.x,b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,b);
    res.z=__fadd_rd(a.z,b);;
  }
  else {
    res.y=__fadd_rd(a.y,b);
    res.z=__fadd_ru(a.z,b);
  }
  res.error=a.error;
  return res;
}


__device__ float_gpu_st operator+(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,b.x);
  else res.x=__fadd_rd(a,b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,b.y);
    res.z=__fadd_rd(a,b.z);;
  }
  else {
    res.y=__fadd_rd(a,b.y);
    res.z=__fadd_ru(a,b.z);
  }
  res.error=b.error;
  return res;
}

__device__ float_gpu_st operator+=(float_gpu_st& a, const float_gpu_st& b)
{
  //float_gpu_st res;
  unsigned char random;

  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  random = RANDOMGPU();
  if (random&1) a.x=__fadd_ru(a.x,b.x);
  else a.x=__fadd_rd(a.x,b.x);

  if (random>>1) {
    a.y=__fadd_ru(a.y,b.y);
    a.z=__fadd_rd(a.z,b.z);;
  }
  else {
    a.y=__fadd_rd(a.y,b.y);
    a.z=__fadd_ru(a.z,b.z);
  }

  a.error=a.error | b.error;

  return a;
}


__device__ float_gpu_st operator+=(float_gpu_st& a, const float& b)
{
  //float_gpu_st res;
  unsigned char random;


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;

  random = RANDOMGPU();
  if (random&1) a.x=__fadd_ru(a.x,b);
  else a.x=__fadd_rd(a.x,b);

  if (random>>1) {
    a.y=__fadd_ru(a.y,b);
    a.z=__fadd_rd(a.z,b);;
  }
  else {
    a.y=__fadd_rd(a.y,b);
    a.z=__fadd_ru(a.z,b);
  }
  //res.error=a.error;
  return a;
}



/////////////////////////////////////////////////////


__device__ float_gpu_st operator-(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }

  res.error= a.error | b.error;
  return res;
}


__device__ float_gpu_st operator-(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;

  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  res.error= a.error;

  return res;

}

__device__ float_gpu_st operator-(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  res.error= b.error;
  return res;
}

__device__ float_gpu_st operator-=(float_gpu_st& a, const float_gpu_st& b)
{
  //float_gpu_st res;
  unsigned char random;


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  random = RANDOMGPU();
  if (random&1) a.x=__fadd_ru(a.x,-b.x);
  else a.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    a.y=__fadd_ru(a.y,-b.y);
    a.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    a.y=__fadd_rd(a.y,-b.y);
    a.z=__fadd_ru(a.z,-b.z);
  }

  a.error= a.error | b.error;
  return a;
}


__device__ float_gpu_st operator-=(float_gpu_st& a, const float& b)
{
  //float_gpu_st res;
  unsigned char random;


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;

  random = RANDOMGPU();
  if (random&1) a.x=__fadd_ru(a.x,-b);
  else a.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    a.y=__fadd_ru(a.y,-b);
    a.z=__fadd_rd(a.z,-b);;
  }
  else {
    a.y=__fadd_rd(a.y,-b);
    a.z=__fadd_ru(a.z,-b);
  }
  //res.error= a.error;

  return a;

}


/////////////////////////////////////////////////////

__device__ float_gpu_st operator*(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;

  unsigned int inst;

  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fmul_ru(a.x,b.x);
  else res.x=__fmul_rd(a.x,b.x);
  if (random>>1) {
    res.y=__fmul_ru(a.y,b.y);
    res.z=__fmul_rd(a.z,b.z);;
  }
  else {
    res.y=__fmul_rd(a.y,b.y);
    res.z=__fmul_ru(a.z,b.z);
  }


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  if (a.accuracy==DIGIT_NOT_COMPUTED	)
    a.approx_digit();
  if (b.accuracy==DIGIT_NOT_COMPUTED)
    b.approx_digit();
  inst=(a.accuracy==0 &&  b.accuracy==0		 ) ? CADNA_MUL : 0;
  res.error=a.error | b.error | inst;
  return res;
}



__device__ float_gpu_st operator*(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fmul_ru(a.x,b);
  else res.x=__fmul_rd(a.x,b);
  if (random>>1) {
    res.y=__fmul_ru(a.y,b);
    res.z=__fmul_rd(a.z,b);;
  }
  else {
    res.y=__fmul_rd(a.y,b);
    res.z=__fmul_ru(a.z,b);
  }
  res.error=a.error;

  return res;
}



__device__ float_gpu_st operator*(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;



  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fmul_ru(a,b.x);
  else res.x=__fmul_rd(a,b.x);
  if (random>>1) {
    res.y=__fmul_ru(a,b.y);
    res.z=__fmul_rd(a,b.z);;
  }
  else {
    res.y=__fmul_rd(a,b.y);
    res.z=__fmul_ru(a,b.z);
  }
  res.error=b.error;
  return res;
}
__device__ float_gpu_st operator*=(float_gpu_st& a, const float_gpu_st& b)
{
  //float_gpu_st res;
  unsigned char random;

  unsigned int inst;

  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  random = RANDOMGPU();
  if (random&1) a.x=__fmul_ru(a.x,b.x);
  else a.x=__fmul_rd(a.x,b.x);
  if (random>>1) {
    a.y=__fmul_ru(a.y,b.y);
    a.z=__fmul_rd(a.z,b.z);;
  }
  else {
    a.y=__fmul_rd(a.y,b.y);
    a.z=__fmul_ru(a.z,b.z);
  }


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  if (a.accuracy==DIGIT_NOT_COMPUTED	)
    a.approx_digit();
  if (b.accuracy==DIGIT_NOT_COMPUTED)
    b.approx_digit();
  inst=(a.accuracy==0 &&  b.accuracy==0		 ) ? CADNA_MUL : 0;
  a.error=a.error | b.error | inst;
  return a;
}



__device__ float_gpu_st operator*=(float_gpu_st& a, const float& b)
{
  //float_gpu_st res;
  unsigned char random;


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  random = RANDOMGPU();
  if (random&1) a.x=__fmul_ru(a.x,b);
  else a.x=__fmul_rd(a.x,b);
  if (random>>1) {
    a.y=__fmul_ru(a.y,b);
    a.z=__fmul_rd(a.z,b);;
  }
  else {
    a.y=__fmul_rd(a.y,b);
    a.z=__fmul_ru(a.z,b);
  }
  //res.error=a.error;

  return a;
}


///////////////////////////////////////////

__device__ float_gpu_st operator/(const float_gpu_st& a, const float_gpu_st& b)
{
  unsigned int inst;
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fdiv_ru(a.x,b.x);
  else res.x=__fdiv_rd(a.x,b.x);
  if (random>>1) {
    res.y=__fdiv_ru(a.y,b.y);
    res.z=__fdiv_rd(a.z,b.z);;
  }
  else {
    res.y=__fdiv_rd(a.y,b.y);
    res.z=__fdiv_ru(a.z,b.z);
  }

  if (b.accuracy==DIGIT_NOT_COMPUTED)
    b.approx_digit();
  inst=( b.accuracy==0    ) ? CADNA_DIV : 0;
  res.error=a.error | b.error | inst;

  return res;
}


__device__ float_gpu_st operator/(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fdiv_ru(a.x,b);
  else res.x=__fdiv_rd(a.x,b);
  if (random>>1) {
    res.y=__fdiv_ru(a.y,b);
    res.z=__fdiv_rd(a.z,b);;
  }
  else {
    res.y=__fdiv_rd(a.y,b);
    res.z=__fdiv_ru(a.z,b);
  }
  res.error=a.error;

  return res;
}


__device__ float_gpu_st operator/(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;

  res.accuracy=DIGIT_NOT_COMPUTED;
  res.error=0;
  random = RANDOMGPU();
  if (random&1) res.x=__fdiv_ru(a,b.x);
  else res.x=__fdiv_rd(a,b.x);
  if (random>>1) {
    res.y=__fdiv_ru(a,b.y);
    res.z=__fdiv_rd(a,b.z);;
  }
  else {
    res.y=__fdiv_rd(a,b.y);
    res.z=__fdiv_ru(a,b.z);
  }

  if (b.accuracy==DIGIT_NOT_COMPUTED)
    b.approx_digit();
  
  res.error=b.error |((b.accuracy==0)  ? CADNA_DIV : 0); //FJ 19 June 2017
  // res.error=b.error |(b.accuracy ? CADNA_DIV : 0); //old
  return res;
}

__device__ float_gpu_st operator/=(float_gpu_st& a, const float_gpu_st& b)
{
  unsigned int inst;
  //float_gpu_st res;
  unsigned char random;


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  random = RANDOMGPU();
  if (random&1) a.x=__fdiv_ru(a.x,b.x);
  else a.x=__fdiv_rd(a.x,b.x);
  if (random>>1) {
    a.y=__fdiv_ru(a.y,b.y);
    a.z=__fdiv_rd(a.z,b.z);;
  }
  else {
    a.y=__fdiv_rd(a.y,b.y);
    a.z=__fdiv_ru(a.z,b.z);
  }

  if (b.accuracy==DIGIT_NOT_COMPUTED)
    b.approx_digit();
  inst=( b.accuracy==0    ) ? CADNA_DIV : 0;
  a.error=a.error | b.error | inst;

  return a;
}


__device__ float_gpu_st operator/=(float_gpu_st& a, const float& b)
{
  //float_gpu_st res;
  unsigned char random;


  //res.accuracy=DIGIT_NOT_COMPUTED;
  //res.error=0;
  random = RANDOMGPU();
  if (random&1) a.x=__fdiv_ru(a.x,b);
  else a.x=__fdiv_rd(a.x,b);
  if (random>>1) {
    a.y=__fdiv_ru(a.y,b);
    a.z=__fdiv_rd(a.z,b);;
  }
  else {
    a.y=__fdiv_rd(a.y,b);
    a.z=__fdiv_ru(a.z,b);
  }
  //res.error=a.error;

  return a;
}


///////////////////////////////////////

__device__ int operator==(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }
  return res.computedzero();
}

__device__ int operator==(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  return res.computedzero();
}

__device__ int operator==(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  return res.computedzero();
}

///////////////////////////////////////

///////////////////////////////////////

__device__ int operator!=(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }
  return !res.computedzero();
}

__device__ int operator!=(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  return !res.computedzero();
}

__device__ int operator!=(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  return !res.computedzero();
}

///////////////////////////////////////

__device__ int operator>(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }
  int r=res.isnumericalnoise();

  if (r) {
    a.error |= CADNA_BRANCHING;
    b.error |= CADNA_BRANCHING;
  }

  return !r && ( ( a.x + a.y + a.z ) >	( b.x + b.y + b.z ));
}


__device__ int operator>(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  int r=res.isnumericalnoise();

  if (r) {
    a.error |= CADNA_BRANCHING;
  }


  return !r && ( ( a.x + a.y + a.z ) > 3*b );
}



__device__ int operator>(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  int r=res.isnumericalnoise();

  if (r) {
    b.error |= CADNA_BRANCHING;
  }


  return !r && ( 3*a  >	( b.x + b.y + b.z ));
}




///////////////////////////////////////


__device__ int operator>=(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }
  int r=res.isnumericalnoise();

  if (r) {
    a.error |= CADNA_BRANCHING;
    b.error |= CADNA_BRANCHING;
  }

  return r || ( ( a.x + a.y + a.z ) >=	( b.x + b.y + b.z ));
}


__device__ int operator>=(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  int r=res.isnumericalnoise();

  if (r) {
    a.error |= CADNA_BRANCHING;
  }

  return r || ( ( a.x + a.y + a.z ) >=	3*b);
}



__device__ int operator>=(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  int r=res.isnumericalnoise();
  if (r) {
    b.error |= CADNA_BRANCHING;
  }

  return r || ( 3*a  >=	( b.x + b.y + b.z ));
}




///////////////////////////////////////

__device__ int operator<(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }
  int r=res.isnumericalnoise();
  if (r) {
    a.error |= CADNA_BRANCHING;
    b.error |= CADNA_BRANCHING;
  }

  return !r && ( ( a.x + a.y + a.z ) <	( b.x + b.y + b.z ));
}


__device__ int operator<(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  int r=res.isnumericalnoise();
  if (r) {
    a.error |= CADNA_BRANCHING;

  }

  return !r && ( ( a.x + a.y + a.z ) < 3*b );
}



__device__ int operator<(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  int r=res.isnumericalnoise();
  if (r) {
    b.error |= CADNA_BRANCHING;
  }


  return !r && ( 3*a  <	( b.x + b.y + b.z ));
}




///////////////////////////////////////


__device__ int operator<=(const float_gpu_st& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b.x);
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b.y);
    res.z=__fadd_rd(a.z,-b.z);;
  }
  else {
    res.y=__fadd_rd(a.y,-b.y);
    res.z=__fadd_ru(a.z,-b.z);
  }
  int r=res.isnumericalnoise();
  if (r) {
    a.error |= CADNA_BRANCHING;
    b.error |= CADNA_BRANCHING;
  }

  return r || ( ( a.x + a.y + a.z ) <=	( b.x + b.y + b.z ));
}


__device__ int operator<=(const float_gpu_st& a, const float& b)
{
  float_gpu_st res;
  unsigned char random;


  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a.x,-b);
  else res.x=__fadd_rd(a.x,-b);

  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);
    res.z=__fadd_rd(a.z,-b);;
  }
  else {
    res.y=__fadd_rd(a.y,-b);
    res.z=__fadd_ru(a.z,-b);
  }
  int r=res.isnumericalnoise();
  if (r) {
    a.error |= CADNA_BRANCHING;
  }

  return r || ( ( a.x + a.y + a.z ) <=	3*b);
}



__device__ int operator<=(const float& a, const float_gpu_st& b)
{
  float_gpu_st res;
  unsigned char random;

  res.accuracy=DIGIT_NOT_COMPUTED;
  random = RANDOMGPU();
  if (random&1) res.x=__fadd_ru(a,-b.x);
  else res.x=__fadd_rd(a,-b.x);

  if (random>>1) {
    res.y=__fadd_ru(a,-b.y);
    res.z=__fadd_rd(a,-b.z);;
  }
  else {
    res.y=__fadd_rd(a,-b.y);
    res.z=__fadd_ru(a,-b.z);
  }
  int r=res.isnumericalnoise();
  if (r) {

    b.error |= CADNA_BRANCHING;
  }

  return r || ( 3*a  <=	( b.x + b.y + b.z ));
}

///////////////////////////////////////
 __device__  float_gpu_st fabsf(const  float_gpu_st& a) 
{ 
   float_gpu_st res; 
   res.x = fabsf(a.x); 
   res.y = fabsf(a.y); 
   res.z = fabsf(a.z); 
   res.accuracy=DIGIT_NOT_COMPUTED; 
   res.error=a.error;
   return(res); 
}

 __device__  float_gpu_st fabs(const  float_gpu_st& a) 
{ 
   float_gpu_st res; 
   res.x = fabs(a.x); 
   res.y = fabs(a.y); 
   res.z = fabs(a.z); 
   res.accuracy=DIGIT_NOT_COMPUTED; 
   res.error=a.error;
   return(res); 
}

 __device__  float_gpu_st sqrtf(const  float_gpu_st& a) 
{ 
  float_gpu_st res; 
  unsigned char random;
  random = RANDOMGPU();
 
  if (random&1) res.x=__fsqrt_ru(a.x);	    
  else res.x=__fsqrt_rd(a.x);

  if (random>>1) {
     res.y=__fsqrt_ru(a.y);				 
     res.z=__fsqrt_rd(a.z);				 
  }							 
  else {							 
    res.y=__fsqrt_rd(a.y);				 
    res.z=__fsqrt_ru(a.z);					
  }				
   res.accuracy=DIGIT_NOT_COMPUTED; 
   res.error=a.error;
   return(res); 
}
///////////////////////////////////////
__device__  float_gpu_st fmaxf(const float& a, const float_gpu_st& b) 
{ 
  float_gpu_st res;   
  unsigned char random;
  random = RANDOMGPU();

  if (random&1) res.x=__fadd_ru(a,-b.x);			 
  else res.x=__fadd_rd(a,-b.x);
  if (random>>1) {						 
    res.y=__fadd_ru(a,-b.y);				 
    res.z=__fadd_rd(a,-b.z);				 
  }					       	 
  else {							 
    res.y=__fadd_rd(a,-b.y);				 
    res.z=__fadd_ru(a,-b.z);					
  }			
  if (res.isnumericalnoise()){
      res.x=a; 
      res.y=a; 
      res.z=a; 
      res.accuracy=7;
      res.error=CADNA_BRANCHING;
  }
  else { 
    if ( 3.f*a > (b.x + b.y + b.z)) {
      res.x=a; 
      res.y=a; 
      res.z=a; 
      res.accuracy=7;
      res.error=0;	
    }
    else							 
      res=b;	
  }
  return(res); 
}


__device__  float_gpu_st fmaxf(const  float_gpu_st& a, const  float& b) 
{ 
  float_gpu_st res;  
  unsigned char random;
  random = RANDOMGPU();

  if (random&1) res.x=__fadd_ru(a.x,-b);			 
  else res.x=__fadd_rd(a.x,-b);
  if  (random>>1) {
    res.y=__fadd_ru(a.y,-b);				 
    res.z=__fadd_rd(a.z,-b);				 
  }					       	 
  else {							 
    res.y=__fadd_rd(a.y,-b);				 
    res.z=__fadd_ru(a.z,-b);					
  }			
  if (res.isnumericalnoise()){
      res.x=b; 
      res.y=b; 
      res.z=b; 
      res.accuracy=7;
      res.error=CADNA_BRANCHING;
  }
  else { 
    if (( a.x + a.y + a.z ) > 3.f*b ) {
	res=a;	
    }
    else {								 
      res.x=b; 
      res.y=b; 
      res.z=b; 
      res.accuracy=7;
      res.error=0;	
    }
  }
  return(res); 
}


__device__  float_gpu_st fmaxf(const  float_gpu_st& a, const  float_gpu_st& b) 
{ 
  float_gpu_st res;  
  unsigned char random;
  random = RANDOMGPU();

  if (random&1) res.x=__fadd_ru(a.x,-b.x);			 
  else res.x=__fadd_rd(a.x,-b.x);				 
  if (random>>1) {						 
    res.y=__fadd_ru(a.y,-b.y);				 
    res.z=__fadd_rd(a.z,-b.z);				 
  }					       	 
  else {							 
    res.y=__fadd_rd(a.y,-b.y);				 
    res.z=__fadd_ru(a.z,-b.z);					
  }			
  if (res.isnumericalnoise()){
    if (a.accuracy==DIGIT_NOT_COMPUTED)
      a.nb_significant_digit();						 
    if (b.accuracy==DIGIT_NOT_COMPUTED)
      b.nb_significant_digit();
    if (a.accuracy > b.accuracy ){					 
	res=a;		
        res.error=a.error|CADNA_BRANCHING;
        }			 							 
    else {			 					 
	res=b;	
        res.error=b.error|CADNA_BRANCHING;
        }
  }
  else { 
    if (( a.x + a.y + a.z ) > ( b.x + b.y + b.z )) {
	res=a;	
    }
    else {								 
	res=b;	
    }
  }
  return(res); 
}

//////

__device__  float_gpu_st fminf(const float& a, float_gpu_st& b) 
{ 
  float_gpu_st res; 
  unsigned char random;
  random = RANDOMGPU();

  if (random&1) res.x=__fadd_ru(a,-b.x);			 
  else res.x=__fadd_rd(a,-b.x);
  if (random>>1) {						 
    res.y=__fadd_ru(a,-b.y);				 
    res.z=__fadd_rd(a,-b.z);				 
  }					       	 
  else {							 
    res.y=__fadd_rd(a,-b.y);				 
    res.z=__fadd_ru(a,-b.z);					
  }			
  if (res.isnumericalnoise()){
      res.x=a; 
      res.y=a; 
      res.z=a; 
      res.accuracy=7;
      res.error=CADNA_BRANCHING;
  }
  else { 
    if ( 3.f*a < (b.x + b.y + b.z)) {
      res.x=a; 
      res.y=a; 
      res.z=a; 
      res.accuracy=7;
      res.error=0;	
    }
    else							 
      res=b;	
  }
  return(res); 
}


__device__  float_gpu_st fminf(const  float_gpu_st& a, const  float& b) 
{ 
  float_gpu_st res;
  unsigned char random;
  random = RANDOMGPU();

  if (random&1) res.x=__fadd_ru(a.x,-b);			 
  else res.x=__fadd_rd(a.x,-b);
  if (random>>1) {
    res.y=__fadd_ru(a.y,-b);				 
    res.z=__fadd_rd(a.z,-b);				 
  }					       	 
  else {							 
    res.y=__fadd_rd(a.y,-b);				 
    res.z=__fadd_ru(a.z,-b);					
  }			
  if (res.isnumericalnoise()){
      res.x=b; 
      res.y=b; 
      res.z=b; 
      res.accuracy=7;
      res.error=CADNA_BRANCHING;
  }
  else { 
    if (( a.x + a.y + a.z ) < 3.f*b ) {
	res=a;	
    }
    else {								 
      res.x=b; 
      res.y=b; 
      res.z=b; 
      res.accuracy=7;
      res.error=0;	
    }
  }
  return(res); 
}



__device__  float_gpu_st fminf(const  float_gpu_st& a, const  float_gpu_st& b) 
{ 
  float_gpu_st res;
  unsigned char random;
  random = RANDOMGPU();

  if (random&1) res.x=__fadd_ru(a.x,-b.x);			 
  else res.x=__fadd_rd(a.x,-b.x);

  if (random>>1) {				 
    res.y=__fadd_ru(a.y,-b.y);				 
    res.z=__fadd_rd(a.z,-b.z);				 
  }					       	 
  else {							 
    res.y=__fadd_rd(a.y,-b.y);				 
    res.z=__fadd_ru(a.z,-b.z);					
  }			
  if (res.isnumericalnoise()){
    if (a.accuracy==DIGIT_NOT_COMPUTED)
      a.nb_significant_digit();						 
    if (b.accuracy==DIGIT_NOT_COMPUTED)
      b.nb_significant_digit();
    if (a.accuracy > b.accuracy ){					 
	res=a;		
        res.error=a.error|CADNA_BRANCHING;
        }			 							 
    else {			 					 
	res=b;	
        res.error=b.error|CADNA_BRANCHING;
        }
  }
  else { 
    if (( a.x + a.y + a.z ) < ( b.x + b.y + b.z )) {
	res=a;	
    }
    else {								 
	res=b;	
    }
  }
  return(res); 
}

///////////////////////////////////////


__device__ void float_gpu_st::modify(const int &a)
{
  accuracy |=a;
}


__device__ float_gpu_st& float_gpu_st::operator=(const float &a)
{
  x=a;
  y=a;
  z=a;
  accuracy=7;
  error=0;
  return *this ;
}

__device__ int  float_gpu_st::nb_significant_digit() const
{
  float x0,x1,x2,xx;

  xx=x+y+z;

  accuracy=0;
  if (xx==0.0){
    if ((x==y) &&(x==z) ) accuracy=7;
  }
  else {
    xx=3/xx;
    x0=x*xx-1;
    x1=y*xx-1;
    x2=z*xx-1;
    //FJ 4 Mar 2014:
    float yy=(x0*x0+x1*x1+x2*x2)*(float)3.08546617;
    if (yy<=1.e-14)  accuracy=7;
    else {
      yy= -log10(yy);
      if (yy>=0.) accuracy=(int)((yy+(float)1.)*(float)0.5);
    }
  }
  return accuracy;
}

