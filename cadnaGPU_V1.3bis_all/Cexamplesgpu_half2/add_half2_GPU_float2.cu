#include <cstdlib>
#include <ctime>
/*#include <stdio.h>
#include <time.h>*/
#include <stdlib.h>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <assert.h>

#ifndef N
#define N 0x10000000
#endif

#ifndef TAILLE_BLOC_X
#define TAILLE_BLOC_X 32
#endif

#ifndef NLOOP
#define NLOOP 1
#endif

#ifndef CPT
#define CPT 100
#endif




#define SEED 1

using namespace std;
/*
// Génération aléatoire du tableau :
__host__ void aleaTab(__half* T, int n){
    long int i;
    srand(SEED);

    //    posix_memalign((void **)&T, 64, N * sizeof(float));
    for(i = 0; i < n; i++){
        T[i] = 0.5 + (__half)rand()/(__half)RAND_MAX;

    }
}
*/
// Mesures :
double my_gettimeofday(){
    struct timeval tmp_time;
    gettimeofday(&tmp_time, NULL);
    return tmp_time.tv_sec + (tmp_time.tv_usec * 1.0e-6L);
}


/*
__global__ void add(float a, float b,float res)
{
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	half a_h = __float2half(a);
	half b_h = __float2half(b);
	half res_h = __hadd(a_h,b_h);
	res = __half2float(res_h);
}
*/

/*
__device__ void add()
{
	half x = __hadd(0.,0.);
	printf("");
}
*/

__device__ void sub()
{

	float2 a,b,c;
       a.x = 1.01f;
       a.y = 2.f; 
	b.x = 3.f;
	b.y = 4.f;
	c = {1,2};
	half2 res_a, res_b,res_c;
	res_a = __float22half2_rn(a);
	res_b = __float22half2_rn(b);
	res_c = __float22half2_rn(c);
	
	__half2 res = __hsub2(res_a, res_c);
	
	printf("res = %f, a = %f\n",__half22float2(res), (float2)a);


}



/*
__device__ void mul()
{
	half z = __hmul(0.,0.);
}

__device__ void div()
{
	half d = __hdiv(0.,0.);
}
*/
/*
__global__ void add_kernel()
{
	add();
}
*/
__global__ void sub_kernel()
{
	sub();
}
/*
__global__ void mul_kernel()
{
	mul();
}

__global__ void div_kernel()
{
	div();
}

*/
int main(int argc, char **argv){
    double t_startGPU, t_endGPU;

  /*  float *a, *b, *res;
    a = (float*)malloc(1*sizeof(float));
    b = (float*)malloc(1*sizeof(float));
    res = (float*)malloc(1*sizeof(float));
    a[0] = (float)0.126;
    b[0] = (float)0.523;
    res[0] = 0;
    float a_h,b_h,res_h;
    cudaMalloc(&a_h,1*sizeof(float));
    cudaMalloc(&b_h,1*sizeof(float));
    cudaMalloc(&res_h,1*sizeof(float));
	cudaMemcpy(a_h,a,1*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(b_h,b,1*sizeof(float),cudaMemcpyHostToDevice);
	cudaMemcpy(res_h,res,1*sizeof(float),cudaMemcpyHostToDevice);
    */
    /* Lancement de kernel (asynchrone) : */
    dim3 threadsParBloc(TAILLE_BLOC_X);


    dim3 tailleGrille(ceil(N/(float) TAILLE_BLOC_X));

 
    t_startGPU = my_gettimeofday();
    //add<<<1,1>>>(a_h,b_h,res_h);
	//add_kernel<<<1,1>>>();
	sub_kernel<<<1,1>>>();
	//mul_kernel<<<1,1>>>();
	//div_kernel<<<1,1>>>();
    hipDeviceSynchronize();
 //   cudaMemcpy(res,res_h,1,cudaMemcpyDeviceToHost);
   
    t_endGPU =  my_gettimeofday();

    	//printf("a =%f, b= %f, res = %f\n",a,b,res);
	printf("Time = %d", t_endGPU - t_startGPU);
	/*cudaFree(a);
	cudaFree(b);
	cudaFree(res);
*/
    return EXIT_SUCCESS;
}
