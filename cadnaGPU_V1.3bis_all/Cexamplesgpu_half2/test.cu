#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_fp16.h>
#define DSIZE 4
#define SCF 0.5f
#define nTPB 256

#include <cadna.h>
#include "cadna_gpu.cu"


__global__ void half_scale_kernel(float *din, float *dout, int dsize){

	  int idx = threadIdx.x+blockDim.x*blockIdx.x;
	    if (idx < dsize){
		half_gpu_st scf = __float2half(SCF);
		half_gpu_st kin = __float2half(din[idx]);
		half_gpu_st kout;
#if __CUDA_ARCH__ >= 530
		kout = __hmul(kin, scf);
#else
		kout = __float2half(__half2float(kin)*__half2float(scf));
#endif
		dout[idx] = __half2float(kout);
		}
}

int main(){
		cadna_init(-1,CADNA_INTRINSIC);
	  float *hin, *hout, *din, *dout;
	  hin  = (float *)malloc(DSIZE*sizeof(float));
	  hout = (float *)malloc(DSIZE*sizeof(float));
     	  for (int i = 0; i < DSIZE; i++) hin[i] = i;
	  hipMalloc(&din,  DSIZE*sizeof(float));
	  hipMalloc(&dout, DSIZE*sizeof(float));
	  hipMemcpy(din, hin, DSIZE*sizeof(float), hipMemcpyHostToDevice);
	  half_scale_kernel<<<(DSIZE+nTPB-1)/nTPB,nTPB>>>(din, dout, DSIZE);
	  hipMemcpy(hout, dout, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
	  for (int i = 0; i < DSIZE; i++) printf("%f\n", hout[i]);
	  cadna_end();
	  return 0;
}
